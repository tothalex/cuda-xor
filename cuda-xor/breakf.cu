#include "hip/hip_runtime.h"
__global__
 void _break(char *text,const int size)
 {
   
   char temp_key[key_size];
   temp_key[0] = blockIdx.x/10 + 48;
   temp_key[1] = blockIdx.x%10 + 48;
   temp_key[2] = blockIdx.y/10 + 48;
   temp_key[3] = blockIdx.y%10 + 48;
   temp_key[4] = blockIdx.z + 48;
   temp_key[5] = threadIdx.x + 48;
   temp_key[6] = threadIdx.y + 48;
   temp_key[7] = threadIdx.z + 48;

   for(int i = 0; i < size; i++)
    {
        switch(text[i] ^ temp_key[i % key_size])
        {
        case '|':
        case '~':
        case '^':
        case '*':
        case '+':
        case '_':
        case '{':
        case '}':
        case '\\':
        case '#':
            return;
        }
    }

    printf("%c%c%c%c%c%c%c%c\n",temp_key[0],temp_key[1],temp_key[2],temp_key[3],temp_key[4]
                                 ,temp_key[5],temp_key[6],temp_key[7]);




 }