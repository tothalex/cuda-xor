#include <iostream>
#include <fstream>
using namespace std;

#define key_size 8
#include "beolvas.cu"
#include "breakf.cu"

int main(int argc, char const *argv[])
{
 
   //read
   string file_name = *(argv+1); 
   int text_size = getsize();
   char* text = new char[text_size];
   betext(text);

   //copy the text to the device
   char *d_text;
   hipMalloc((void**)&d_text,text_size);
   hipMemcpy(d_text,text,text_size,hipMemcpyHostToDevice);

   //set the block/threads
   dim3 BlocksPerGrid(100,100,10);
   dim3 ThreadsPerBlock(10,10,10);

   _break<<<BlocksPerGrid,ThreadsPerBlock>>>(d_text,text_size);
    
   hipFree(d_text);
   delete[] text;   
 return 0;
}